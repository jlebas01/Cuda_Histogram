#include "hip/hip_runtime.h"
//
// Created by jlebas01 on 04/04/2020.
//

#include <kernels/kernel.hpp>


namespace kernel {

    __device__ float4 fHSV_from_RGB(float r, float g, float b) {
        float M = 0.0f, m = 0.0f, c = 0.0f;
        float4 HSVcolor = make_float4(0.f, 0.f, 0.f, 255.0f); //x : Hue, y : Saturation, z : Value, w : Opacity
        M = fmax(r, fmax(g, b));
        m = fmin(r, fmin(g, b));
        c = M - m;
        HSVcolor.z = M;
        if (c != 0.0f) {
            if (M == r) {
                HSVcolor.x = fmod(((g - b) / c), 6.0f);
            } else if (M == g) {
                HSVcolor.x = (b - r) / c + 2.0f;
            } else /*if(M==b)*/
            {
                HSVcolor.x = (r - g) / c + 4.0f;
            }
            HSVcolor.x *= 60.0f;
            HSVcolor.y = c / HSVcolor.z;
        }
        //}
        return HSVcolor;
    }

    __device__ float4 fRGB_from_HSV(float h, float s, float v) {
        float c = 0.0f, m = 0.0f, x = 0.0f;
        float4 color = make_float4(0.f, 0.f, 0.f, 1.0f);
        // if (Hsv_IsValid(h, s, v) == true) {
        c = v * s;
        x = c * (1.0f - fabs(fmod(h / 60.0f, 2.0f) - 1.0f));
        m = v - c;
        if (h >= 0.0f && h < 60.0) {
            color = make_float4(c + m, x + m, m, 1.0f);
        } else if (h >= 60.0f && h < 120.0f) {
            color = make_float4(x + m, c + m, m, 1.0f);
        } else if (h >= 120.0f && h < 180.0f) {
            color = make_float4(m, c + m, x + m, 1.0f);
        } else if (h >= 180.0f && h < 240.0f) {
            color = make_float4(m, x + m, c + m, 1.0f);
        } else if (h >= 240.0f && h < 300.0f) {
            color = make_float4(x + m, m, c + m, 1.0f);
        } else if (h >= 300.0f && h < 360.0f) {
            color = make_float4(c + m, m, x + m, 1.0f);
        } else {
            color = make_float4(m, m, m, 1.0f);
        }
        //  }
        return color;
    }

    __device__ float clip(float n, float lower, float upper) {
        return fmax(lower, fmin(n, upper));
    }

    __device__ float4 normalizeRGB(float r, float g, float b, float o) {
        float4 RGBColorNormalized = make_float4(r / 255.f, g / 255.f, b / 255.f, o / 255.f);
        return RGBColorNormalized;
    }

    __global__ void
    normalizePixel(const size_t imgWidth, const size_t imgHeight, float4 *output) {
        uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x);
        uint32_t idy = (blockIdx.y * blockDim.y + threadIdx.y);
        //uchar4 imgInput = make_uchar4(0.0f, 0.0f, 0.0f, 0.0f);

        float4 RGBcolorNomalized = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        if (idx < imgWidth && idy < imgHeight) {

            //uchar4 imgInput =  tex2D(texInput, idx, idy);

            uchar4 imgInput = make_uchar4(0.0f, 0.0f, 0.0f, 0.0f);

            RGBcolorNomalized = normalizeRGB(imgInput.x, imgInput.y, imgInput.z, imgInput.w);

            const uint32_t idOut = idy * imgWidth + idx;
            output[idOut].x = static_cast<uint8_t>(RGBcolorNomalized.x);
            output[idOut].y = static_cast<uint8_t>(RGBcolorNomalized.y);
            output[idOut].z = static_cast<uint8_t>(RGBcolorNomalized.z);
            output[idOut].w = static_cast<uint8_t>(RGBcolorNomalized.w);
        }
    }

    __global__ void
    RGB_to_HSV(const size_t imgWidth, const size_t imgHeight, float4 *output) {
        uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x);
        uint32_t idy = (blockIdx.y * blockDim.y + threadIdx.y);

        float4 HSVColor = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        if (idx < imgWidth && idy < imgHeight) {

            //float4 imgNormalized = tex2D(ImgNormalized, idx, idy);

            float4 imgNormalized = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

            HSVColor = fHSV_from_RGB(imgNormalized.x, imgNormalized.y, imgNormalized.z);

            const uint32_t idOut = idy * imgWidth + idx;
            output[idOut].x = static_cast<float>(HSVColor.x);
            output[idOut].y = static_cast<float>(HSVColor.y);
            output[idOut].z = static_cast<float>(HSVColor.z);
            output[idOut].w = static_cast<float>(HSVColor.w);
        }
    }

    __global__ void
    HSV_to_RGB(const size_t imgWidth, const size_t imgHeight, uchar4 *output) {
        uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x);
        uint32_t idy = (blockIdx.y * blockDim.y + threadIdx.y);

        float4 RGBColor = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        if (idx < imgWidth && idy < imgHeight) {

            //float4 imgHSV = tex2D<float4>(ImgHSV, idx, idy);

            float4 imgHSV =make_float4(0.0f, 0.0f, 0.0f, 0.0f);

            RGBColor = fRGB_from_HSV(imgHSV.x, imgHSV.y, imgHSV.z);

            const uint32_t idOut = idy * imgWidth + idx;
            output[idOut].x = static_cast<uint8_t>(RGBColor.x*255.f);
            output[idOut].y = static_cast<uint8_t>(RGBColor.y*255.f);
            output[idOut].z = static_cast<uint8_t>(RGBColor.z*255.f);
            output[idOut].w = static_cast<uint8_t>(RGBColor.w*255.f);
        }
    }

}